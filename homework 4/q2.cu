#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define bzero(b,len) (memset((b), '\0', (len)), (void) 0)


__global__ void globalCount(int *count, int *d_in, int size)
{	
	int myId = threadIdx.x + blockDim.x * blockIdx.x;
	int tId = threadIdx.x;
	if (tId < size) {
		count[d_in[tId] / 10]++;
	}
}

__global__ void sharedCount(int *count, int *d_in, int size)
{	
	etern __shared__ int sCount[10];

	int myId = threadIdx.x + blockDim.x * blockIdx.x;
	int tId = threadIdx.x;

	sCount[tId] = count[tId];
	__syncthreads();

	//counts in shared memory
	if (tId < size) {
		sCount[d_in[tId]/10]++;
	}
	__syncthreads();
	//writes back to global memory
	if (tId < size) {
		count[tId] += sCount[d_in[tId]/10];
	}
}

__global__ void prefixCount(int *count, int *d_in, int size) {
	int myId = threadIdx.x + blockDim.x * blockIdx.x;
	int tId = threadIdx.x;
	if (tId < size) {
		for (int i = 0; i < tId; i++) {
			count[tId] += d_in[i];
		}
		__syncthreads();
	}
}

void reduce(int *count3, int *count2, int *count1, int *d_in, int size) {
	const int maxThreadsPerBlock = 512;
	int threads = maxThreadsPerBlock;
	int blocks;
	if (size > maxThreadsPerBlock) {
		int blocks = size / maxThreadsPerBlock;
	}
	else {
		blocks = 1;
	}
	
	globalCount(count1, d_in, size);
	sharedCount(count2, d_in, size);
	prefixCount(count3, count1, 10);
}

int main()
{
	const int Array_Size = 1000;
	const int Array_Bytes = Array_Size * sizeof(int);
	const int Count_Size = 10;
	const int Count_Bytes = Count_Size * sizeof(int);

	int num[Array_Size];
	FILE *ifp, *ofp1, *ofp2, *ofp3;
	char ch, buffer[32];
	int i = 0, j = 0;

	//Open the file
	ifp = fopen("inp.txt", "r");
	if (ifp == NULL) {
		fprintf(stderr, "Can't open input file inp.txt");
		return 0;
	}

	//Change this!
	ofp1 = fopen("q2a.txt", "w");
	ofp2 = fopen("q2b.txt", "w");
	ofp3 = fopen("q2c.txt", "w");

	//Take number and put into num array
	while (1) {
		ch = fgetc(ifp);
		if (ch == EOF) {
			if (ch == ',') {
				break;
			}
			else {
				int x = atoi(buffer);
				if (x < 1000 && x >= 0) {
					num[j] = atoi(buffer);
					j++;
				}
				bzero(buffer, 32);
				i = 0;
				break;
			}
		}
		else if (ch == ',') {
			int x = atoi(buffer);
			if (x < 1000 && x >= 0) {
				num[j] = atoi(buffer);
				j++;
			}
			bzero(buffer, 32);
			i = 0;
			continue;
		}
		else {
			buffer[i] = ch;
			i++;
		}
	}

	int zeros[] = { 0,0,0,0,0,0,0,0,0,0 };
	int *d_in, *count1, *count2, *count3;

	hipMalloc((void**)&d_in, Array_Bytes);
	hipMalloc((void**)&count1, Array_Bytes);
	hipMalloc((void**)&count2, Array_Bytes);
	hipMalloc((void**)&count3, Array_Bytes);

	//Allocate Memory
	hipMemcpy(d_in, num, Array_Bytes, hipMemcpyHostToDevice);
	hipMemcpy(count1, zeros, Count_Bytes, hipMemcpyHostToDevice);
	hipMemcpy(count2, zeros, Count_Bytes, hipMemcpyHostToDevice);
	hipMemcpy(count3, zeros, Count_Bytes, hipMemcpyHostToDevice);

	reduce(count3, count2, count1, d_in, j);

	//Copy from Device to Host
	int globalCount[Count_Size];
	hipMemcpy(globalCount, count1, Count_Bytes, hipMemcpyDeviceToHost);
	int sharedCount[Count_Size];
	hipMemcpy(sharedCount, count2, Count_Bytes, hipMemcpyDeviceToHost);
	int prefixCount[Count_Size];
	hipMemcpy(prefixCount, count3, Count_Bytes, hipMemcpyDeviceToHost);

	//Output to text file
	for (int i = 0; i < 10; i++) {
		fprintf(ofp1, "%d,", count1[i]);
		fprintf(ofp2, "%d,", count2[i]);
		fprintf(ofp3, "%d,", count3[i]);
	}

	//Close files
	fclose(ifp);
	fclose(ofp1);
	fclose(ofp2);
	fclose(ofp3);

	//Free memory
	hipFree(d_in);
	hipFree(count1);
	hipFree(count2);
	hipFree(count3);

	return 0;
}

