
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define bzero(b,len) (memset((b), '\0', (len)), (void) 0)


__global__ void findMin(int *d_in, int size) 
{
	int tId = threadIdx.x;
	if (tId < size) {
		if (d_in[tId] < d_in[0]) {
			d_in[0] = d_in[tId];
		}
	}
}

__global__ void firstDigit(int *d_out, int *d_in, int size)
{
	int tId = threadIdx.x % size;
	if (tId < size) {
		d_out[tId] = d_in[tId] % 10;
	}
}

void reduce(int *d_out, int *d_in, int size) {
	const int maxThreadsPerBlock = 512;
	int threads = maxThreadsPerBlock;
	int blocks;
	if (size > maxThreadsPerBlock) {
		int blocks = size / maxThreadsPerBlock;
	}
	else {
		blocks = 1;
	}
	firstDigit <<<blocks, threads >>> (d_out, d_in, size);
	findMin<<<blocks, threads >>>(d_in, size);
}

int main()
{
	const int Array_Size = 1000;
	const int Array_Bytes = Array_Size * sizeof(int);

	int num[Array_Size];
	FILE *ifp, *ofp1, *ofp2;
	char ch, buffer[32];
	int i = 0, j = 0;

	//Open the file
	ifp = fopen("inp.txt", "r");
	if (ifp == NULL) {
		fprintf(stderr, "Can't open input file inp.txt");
		return 0;
	}

	//Change this!
	ofp1 = fopen("q1a.txt", "w");
	ofp2 = fopen("q1b.txt", "w");

	//Take number and put into num array
	while (1) {
		ch = fgetc(ifp);
		if (ch == EOF) {
			if (ch == ',') {
				break;
			}
			else {
				int x = atoi(buffer);
				if (x < 1000 && x >= 0) {
					num[j] = atoi(buffer);
					j++;
				}
				bzero(buffer, 32);
				i = 0;
				break;
			}
		}
		else if (ch == ',') {
			int x = atoi(buffer);
			if (x < 1000 && x >= 0) {
				num[j] = atoi(buffer);
				j++;
			}
			bzero(buffer, 32);
			i = 0;
			continue;
		}
		else {
			buffer[i] = ch;
			i++;
		}
	}

	int *d_in, *d_out;

	hipMalloc((void**)&d_in, Array_Bytes);
	hipMalloc((void**)&d_out, Array_Bytes);

	hipMemcpy(d_in, num, Array_Bytes, hipMemcpyHostToDevice);

	reduce(d_out, d_in, j);

	int min;
	hipMemcpy(&min, d_in, sizeof(int), hipMemcpyDeviceToHost);
	int first[Array_Size];
	hipMemcpy(first, d_out, Array_Bytes, hipMemcpyDeviceToHost);

	//Output min to text file
	fprintf(ofp1, "%d", min);
	for (int i = 0; i < j; i++) {
		fprintf(ofp2, "%d,", first[i]);
	}

	//Close files
	fclose(ifp);
	fclose(ofp1);
	fclose(ofp2);

	//Free memory
	hipFree(d_in);
	hipFree(d_out);

    return 0;
}

