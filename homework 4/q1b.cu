
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define bzero(b,len) (memset((b), '\0', (len)), (void) 0)


__global__ void firstDigit(int *d_out, int *d_in, int size)
{
	//int myId = threadIdx.x + blockIdx.x*blockDim.x;
	int tId = threadIdx.x % size;
//	if (tId < size) {
	d_out[tId] = d_in[tId] % 10;
	d_in[tId] = d_in[tId] % 10;
			//d_out[tId] = d_in[tId] - ((d_in[tId]/10)*10);
			
	
		//else {
		//	d_out[tId] = d_in[tId];
		//}
//	}
}

void reduce(int *d_out, int *d_in, int size) {
	const int maxThreadsPerBlock = 512;
	int threads = maxThreadsPerBlock;
	int blocks = size / maxThreadsPerBlock;

	firstDigit<<<blocks, threads >>>(d_out, d_in, size);
}

int main()
{
	const int Array_Size = 1000;
	const int Array_Bytes = Array_Size * sizeof(int);

	int num[Array_Size];
	FILE *ifp, *ofp;
	char ch, buffer[32];
	int i = 0, j = 0;

	//Open the file
	ifp = fopen("inp.txt", "r");
	if (ifp == NULL) {
		fprintf(stderr, "Can't open input file inp.txt");
		return 0;
	}

	//Change this!
	ofp = fopen("q1b.txt", "w");

	//Take number and put into num array
	while (1) {
		ch = fgetc(ifp);
		if (ch == EOF) {
			if (ch == ',') {
				break;
			}
			else {
				int x = atoi(buffer);
				if (x < 1000 && x >= 0) {
					num[j] = atoi(buffer);
					j++;
				}
				bzero(buffer, 32);
				i = 0;
				break;
			}
		}
		else if (ch == ',') {
			int x = atoi(buffer);
			if (x < 1000 && x >= 0) {
				num[j] = atoi(buffer);
				j++;
			}
			bzero(buffer, 32);
			i = 0;
			continue;
		}
		else {
			buffer[i] = ch;
			i++;
		}
	}

	int *d_in, *d_out;

	hipMalloc((void**)&d_in, Array_Bytes);
	hipMalloc((void**)&d_out, Array_Bytes);

	hipMemcpy(d_in, num, Array_Bytes, hipMemcpyHostToDevice);

	reduce(d_out, d_in, j);
	
	int first1[Array_Size];
	int first2[Array_Size];
	hipMemcpy(first1, d_in, Array_Size, hipMemcpyDeviceToHost);
	hipMemcpy(first2, d_out, Array_Size, hipMemcpyDeviceToHost);

	//Output to txt file
	for (int i = 0; i < j; i++) {
		fprintf(ofp, "%d,", first1[i]);
	}
	fprintf(ofp, "\n");
	for (int i = 0; i < j; i++) {
		fprintf(ofp, "%d,", first2[i]);
	}

	//Close files
	fclose(ifp);
	fclose(ofp);

	//Free memory
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}

