
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define bzero(b,len) (memset((b), '\0', (len)), (void) 0)


__global__ void findMin(int *d_in, int size) 
{
	int tId = threadIdx.x;
	if (tId < size) {
		if (d_in[tId] < d_in[0]) {
			d_in[0] = d_in[tId];
		}
	}
	__syncthreads();
}

void reduce(int *d_in, int size) {
	const int maxThreadsPerBlock = 512;
	int threads = maxThreadsPerBlock;
	int blocks;
	if (size > maxThreadsPerBlock) {
		int blocks = size / maxThreadsPerBlock;
	}
	else {
		blocks = 1;
	}
	findMin<<<blocks, threads >>>(d_in, size);
}

int main()
{
	const int Array_Size = 1000;
	const int Array_Bytes = Array_Size * sizeof(int);

	int num[Array_Size];
	FILE *ifp, *ofp;
	char ch, buffer[32];
	int i = 0, j = 0;

	//Open the file
	ifp = fopen("inp.txt", "r");
	if (ifp == NULL) {
		fprintf(stderr, "Can't open input file inp.txt");
		return 0;
	}

	//Change this!
	ofp = fopen("q1a.txt", "w");

	//Take number and put into num array
	while (1) {
		ch = fgetc(ifp);
		if (ch == EOF) {
			if (ch == ',') {
				break;
			}
			else {
				int x = atoi(buffer);
				if (x < 1000 && x >= 0) {
					num[j] = atoi(buffer);
					j++;
				}
				bzero(buffer, 32);
				i = 0;
				break;
			}
		}
		else if (ch == ',') {
			int x = atoi(buffer);
			if (x < 1000 && x >= 0) {
				num[j] = atoi(buffer);
				j++;
			}
			bzero(buffer, 32);
			i = 0;
			continue;
		}
		else {
			buffer[i] = ch;
			i++;
		}
	}

	int *d_in;//, *d_out;

	hipMalloc((void**)&d_in, Array_Bytes);
	//cudaMalloc((void**)&d_out, Array_Bytes);

	hipMemcpy(d_in, num, Array_Bytes, hipMemcpyHostToDevice);

	//reduce(d_out, d_in, j);
	reduce(d_in, j);

	int min;
	hipMemcpy(&min, d_in, sizeof(int), hipMemcpyDeviceToHost);
	//int first[Array_Size];
	//cudaMemcpy(first, d_out, Array_Bytes, cudaMemcpyDeviceToHost);

	//Output min to text file
	fprintf(ofp, "%d\n", min);

	//Output first digit of each number
	//for (int i = 0; i < j; i++) {
	//	fprintf(ofp, "%d,", first[i]);
	//}

	//Close files
	fclose(ifp);
	fclose(ofp);

	//Free memory
	hipFree(d_in);
	//cudaFree(d_out);

    return 0;
}

